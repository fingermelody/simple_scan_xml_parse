#include "hip/hip_runtime.h"
#include "synchronize.h"
#include "simple_state_machine.h"
#include "stack.h"
#include "tag.h"
#include "node.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include "simple_scan_parser.h"
#include <time.h>
#include <pthread.h>
#include <unistd.h>
#include <mpi.h>
#include "Text.h"
#include "tag.h"
#include "tools/hash_map.h"
#include "MPQ_hash.h"
#include "slaves.h"
#include "MPI_MSG_TYPE.h"
#include "debug.h"

#define TagNameAddChar(tag, c) do{			\
	tag->name[tag->nameCharIndex] = c;		\
	tag->nameCharIndex++;					\
}while(0)

#define TagAddAttr(tag, attr) do{			\
	tag->attributes[tag->attrIndex]=attr;	\
	tag->attrIndex++;						\
}while(0)

#define AttrNameAddChar(attr, c) do{		\
	attr->name[attr->nameCharIndex] = c;    \
	attr->nameCharIndex++;					\
}while(0)

#define AttrValueAddChar(attr,c) do{		\
	attr->value[attr->valueCharIndex] = c;  \
	attr->valueCharIndex++;					\
}while(0)

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LIN-E__ ))

//__constant__ unsigned char pBuffer[16*16*16*16*LENGTH];

//static void HandleError( hipError_t err,
//                         const char *file,
//                         int line ) {
//    if (err != hipSuccess) {
//        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
//                file, line );
//        exit( EXIT_FAILURE );
//    }
//}

typedef struct{
	char **host_read_string;
	Tag **host_tags_info;
}cuda_arg;

extern "C"{
	void syn_init();
	void syn_suspend(pthread_cond_t cond);
	void syn_resume(pthread_cond_t cond);
	void* simple_parse(void* arg);
	void hash_map_init(hash_map *map, size_t capacity, hash_map_comparator comparator, hash_map_hash_func hash_func);
	void hash_map_insert(hash_map *map, void *key, void *value);
	void prepareCryptTable();
	size_t hash_map_MPQ_hash_func(const void *key, size_t capacity);
	int MPQ_comparator(const void* l, const void *r);
	void* slave_query_worker(void* argc);
	void* query(void* argc);
	void* schedule(void* argc);
	void* monitor(void* argc);
}

/*
 * GPU kernel. parse the xml file to tags' detail
 * @param: tags_info, tags' basic structure information, this rountine will complete the information in this Tag list.
 * @param: chars of partial xml file got from master.
 * */
__global__ void parse_tag(Tag* tags_info, char* string){
	int id = threadIdx.x;
	Tag t = tags_info[id];
	int s_location = t.location;
	int len = t.lengh;
	
	int curIndex = s_location;
	enum _state st = st_idle;
	attribute tmp_attr;
	while(curIndex - s_location < len){
		char cur = string[curIndex];

		switch(st){
			case st_idle:{
				//start state
				if('<' == cur){
					st = st_lt;
					break;
				}
				break;
			}//end of case st_idle
			case st_lt:{	
				if('>' == cur){
					st = st_idle;
					break;
				}
				//if cur is other character, this node is a start tag
				st = st_start_tag;
				break;
			}//end of state tagstart
			case st_start_tag:{//record tag name here
				if(' ' == cur){
					memset(&tmp_attr,0,sizeof(attribute));
					st = st_attribute_pre;
				}else if('>' == cur){
					st = st_idle;
				}else{// else this char belongs to tag name
					TagNameAddChar((&t),cur);
				}
				break;
			}//end of state st_start_tag
			case st_attribute_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){// normal end
					st = st_idle;
				}else{
					AttrNameAddChar((&t),cur);
					st = st_attribute_name;
				}
				break;
			}
			case st_attribute_name:{
				if('>' == cur){
					st = st_idle;
				}else if('=' == cur){
					st = st_attribute_value_pre;
				}else{
					AttrNameAddChar((&t),cur);
				}
				break;
			}
			case st_attribute_value_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){
					st = st_idle;
				}else if(('"'==cur)||('\''==cur)){
					st = st_attribute_value;
				}
				break;
			}
			case st_attribute_value:{
				if(('"'==cur)||('\''==cur)){
					TagAddAttr((&t),tmp_attr);
					//free(tmp_attr);
					st = st_attribute_pre;
				}else{
					AttrValueAddChar((&tmp_attr),cur);
				}
				break;
			}

		}//end of switch
		
		curIndex ++;
		continue;
	}//end of read buffer.

}


/*
 * slave thread will receive the partial XML file from master, and parse the xml into tags details,
 * including tag name, attribute,then uses these details to build tag invert index, which includes
 * the tag name as the key and the tag id as the value.
 * */
void* slave_parse(void* argc){

	MPI_Status status;
	size_t inverted_index_map_size = 134217728; //2^27
	tags_inverted_index = (hash_map*)malloc(sizeof(hash_map));
	hash_map_init(tags_inverted_index,inverted_index_map_size,MPQ_comparator,hash_map_MPQ_hash_func);
	prepareCryptTable();//CryptTable is for the MPQ hash algorithm
	char* host_read_string; //buffer for receiving the chars of the partial xml file
	host_read_string = (char*)malloc(sizeof(char)*MAX_TEXT_LENGTH);
	memset(host_read_string,0,sizeof(char)*MAX_TEXT_LENGTH);
	char* tag_info_bytes; //buffer for receiving Tags' information in the partial xml file
	tag_info_bytes =(char*)malloc(TAGS_PER_TIME*sizeof(Tag));
	memset(tag_info_bytes,0,sizeof(Tag)*TAGS_PER_TIME);
	int slave_rank;
	MPI_Comm_rank(MPI_COMM_WORLD,&slave_rank);
	printf("%d slave thread start \n",slave_rank);
	Tag* host_tags_buffer = (Tag*)malloc(sizeof(Tag)*TAGS_PER_TIME);//buffer for restoring the tag parse result.
	memset(host_tags_buffer,0,sizeof(Tag)*TAGS_PER_TIME);
	while(1){
//		MPI_Probe(0,MPI_ANY_TAG,MPI_COMM_WORLD,&status);
//		printf("slave receive a message %d\n",status.MPI_TAG);
		int test = 0;
		MPI_Recv(&test,1,MPI_INT,0,MSG_TEST,MPI_COMM_WORLD,&status);
		printf("test is %d \n",test);
		//receive the partial xml file and the structure informations of tags in this part of file
//		MPI_Recv(host_read_string,10240000,MPI_CHAR,0,MSG_SEND_STRING,MPI_COMM_WORLD,&status);
//		MPI_Recv(tag_info_bytes,TAGS_PER_TIME*sizeof(Tag),MPI_CHAR,0,MSG_SEND_TAG_INFO,MPI_COMM_WORLD,&status);
		printf("slave %d receive successfully\n, %s",slave_rank,host_read_string);
//		Tag *host_tags_info = (Tag*)tag_info_bytes;
//		memset(host_tags_buffer,0,TAGS_PER_TIME*sizeof(Tag));
//
//		char* device_string_to_parse;
//		Tag* device_tags_info;
//		size_t host_string_len = strlen(host_read_string);
//
//		hipMalloc((void**)&device_string_to_parse,sizeof(char)*host_string_len);
//		hipMalloc((void**)&device_tags_info,sizeof(Tag)*TAGS_PER_TIME);
//
//		hipMemcpy(device_string_to_parse,host_read_string,host_string_len,hipMemcpyHostToDevice);
//		hipMemcpy(device_tags_info,host_tags_info,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyHostToDevice);
//
//		parse_tag<<<1,1024>>>(device_tags_info, device_string_to_parse);
//
//		hipMemcpy(&host_tags_buffer,device_tags_info,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyDeviceToHost);
//		hipFree(device_string_to_parse);
//		hipFree(device_tags_info);
//		idle = 1;
//		for(int i=0;(i<TAGS_PER_TIME)&&(host_tags_buffer[i].nameCharIndex>0);i++){
//			char* key = host_tags_buffer[i].name;
//			int id = host_tags_buffer[i].id;
//			hash_map_insert(tags_inverted_index,(void*)key,(void*)(&id));
//		}
//		//
//		MPI_Probe(0,MPI_ANY_TAG,MPI_COMM_WORLD,&status);
//		if(status.MPI_TAG == MSG_EXIT)
//			break;
	}
//	pthread_t slave_query_thread;
////	int* temp_arg;
//	pthread_create(&slave_query_thread,NULL,slave_query_worker,NULL);
//	pthread_join(slave_query_thread,NULL);
//	return NULL;


}


char processor_name[MPI_MAX_PROCESSOR_NAME];

int main(int argc, char **argv) {
	char *filePath = argv[1];
	simple_parse_arg *s_arg;

	int numprocs,namelen,rank,devCount;
	MPI_Init(&argc,&argv);
	MPI_Comm_size(MPI_COMM_WORLD,&numprocs);
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Get_processor_name(processor_name,&namelen);
	printf("Hello from %d on %s out of %d\n", rank, processor_name, numprocs);
	slaves_num = numprocs - 1;
	for(int i=0;i<5;i++)
			idle_node[i] = 1;
	printf("set slaves num %d \n",slaves_num);
	if(rank==0){
//		GDB_WAIT_ATTACH();
		//initial values
		file_read_over = 0;
		slave_parse_stop = 0;
		s_arg = (simple_parse_arg*)malloc(sizeof(simple_parse_arg));
		s_arg->file_path = filePath;
		s_arg->s_tags_ready = (Tag**)malloc(sizeof(Tag*));
		s_arg->string_read = (char**)malloc(sizeof(char*));
		*(s_arg->s_tags_ready) = (Tag*)malloc(sizeof(Tag));
		*(s_arg->string_read) = (char*)malloc(sizeof(char));
		pthread_t thread_prescan, thread_schedule, thread_monitor;
		pthread_create(&thread_prescan,NULL,simple_parse,(void*)s_arg);
		pthread_create(&thread_schedule,NULL,schedule,(void*)s_arg);
		pthread_create(&thread_monitor,NULL,monitor,NULL);
		pthread_join(thread_prescan,NULL);
		pthread_join(thread_schedule,NULL);
	}
	else{
		if(hipGetDeviceCount(&devCount)!=hipSuccess){
			 printf("Device error on %s\n", processor_name);
			 MPI_Finalize();
			 return EXIT_FAILURE;
		}

//		GDB_WAIT_ATTACH();
//		int* temp_arg;
		pthread_t slave_parse_thread;
		pthread_create(&slave_parse_thread,NULL,slave_parse,NULL);
		pthread_join(slave_parse_thread,NULL);


	}

	MPI_Finalize();
	return 0;
}
