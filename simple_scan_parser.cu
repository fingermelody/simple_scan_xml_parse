#include "hip/hip_runtime.h"
#include "synchronize.h"
#include "simple_state_machine.h"
#include "stack.h"
#include "tag.h"
#include "node.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include "simple_scan_parser.h"
#include <time.h>
#include <pthread.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <mpi.h>
#include "Text.h"
#include "tag.h"
#include "tools/hash_map.h"
#include "MPQ_hash.h"
#include "slaves.h"
#include "MPI_MSG_TYPE.h"
#include "debug.h"

#define TagNameAddChar(tag, c) do{			\
	tag->name[tag->nameCharIndex] = c;		\
	tag->nameCharIndex++;					\
}while(0)

#define TagAddAttr(tag, attr) do{			\
	tag->attributes[tag->attrIndex]=attr;	\
	tag->attrIndex++;						\
}while(0)

#define AttrNameAddChar(attr, c) do{		\
	attr->name[attr->nameCharIndex] = c;    \
	attr->nameCharIndex++;					\
}while(0)

#define AttrValueAddChar(attr,c) do{		\
	attr->value[attr->valueCharIndex] = c;  \
	attr->valueCharIndex++;					\
}while(0)

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//__constant__ unsigned char pBuffer[16*16*16*16*LENGTH];

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
char file_path[100];
typedef struct{
	char **host_read_string;
	Tag **host_tags_info;
}cuda_arg;

extern "C"{
	void syn_init();
	void syn_suspend(pthread_cond_t cond);
	void syn_resume(pthread_cond_t cond);
	void* simple_parse(void* arg);
	void hash_map_init(hash_map *map, size_t capacity, hash_map_comparator comparator, hash_map_hash_func hash_func);
	void hash_map_insert(hash_map *map, void *key, void *value);
	void prepareCryptTable();
	size_t hash_map_MPQ_hash_func(const void *key, size_t capacity);
	int MPQ_comparator(const void* l, const void *r);
	void* slave_query_worker(void* argc);
	void* query(void* argc);
	void* schedule(void* argc);
	void* monitor_slaves(void* argc);
	void text_init(Text* text);
	void text_add_char(Text* text, char c);
}

__global__ void gpu_test(tag_info* infos,char* string, int* ids,Tag* tags){
	int id = threadIdx.x;
	ids[id] = infos[id].id;
	Tag* t = (&tags[id]);

}

/*
 * GPU kernel. parse the xml file to tags' detail
 * @param: tags_info, tags' basic structure information, this rountine will complete the information in this Tag list.
 * @param: chars of partial xml file got from master.
 * */
__global__ void parse_tag(tag_info* tags_info, char* s, int base, Tag* tags){

	int id = threadIdx.x;
	tag_info info = tags_info[id];

	Tag* t = &(tags[id]);
	int s_location = info.location;
	int len = info.lengh;
	if(len == 0) return;

	t->info = info;



	int curIndex = s_location;
	int end = curIndex + len;
	if(s[curIndex]=='<')
		t->name[0] = 'R';
	else
		t->name[0] = 'E';
//	TagNameAddChar(t,string[curIndex]);

	enum _state st = st_idle;
	attribute tmp_attr;



	while(curIndex  < end){
		char cur = s[curIndex];
		switch(st){
			case st_idle:{
				//start state
				if('<' == cur){
					st = st_lt;
					break;
				}
				break;
			}//end of case st_idle
			case st_lt:{
				if('>' == cur){
					st = st_idle;
					break;
				}
				//if cur is other character, this node is a start tag
				TagNameAddChar(t,cur);
				st = st_start_tag;
				break;
			}
			case st_start_tag:{//record tag name here
				if(' ' == cur){
					memset(&tmp_attr,0,sizeof(attribute));
					st = st_attribute_pre;
				}else if('>' == cur){
					st = st_idle;
				}else{// else this char belongs to tag name
					TagNameAddChar(t,cur);
				}
				break;
			}//end of state st_start_tag
			case st_attribute_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){// normal end
					st = st_idle;
				}else{
					AttrNameAddChar((&tmp_attr),cur);
					st = st_attribute_name;
				}
				break;
			}
			case st_attribute_name:{
				if('>' == cur){
					st = st_idle;
				}else if('=' == cur){
					st = st_attribute_value_pre;
				}else{
					AttrNameAddChar((&tmp_attr),cur);
				}
				break;
			}
			case st_attribute_value_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){
					st = st_idle;
				}else if(('"'==cur)||('\''==cur)){
					st = st_attribute_value;
				}
				break;
			}
			case st_attribute_value:{
				if(('"'==cur)||('\''==cur)){
					TagAddAttr(t,tmp_attr);
					//free(tmp_attr);
					st = st_attribute_pre;
				}else{
					AttrValueAddChar((&tmp_attr),cur);
				}
				break;
			}

		}//end of switch
		
		curIndex ++;
	}//end of while.

	__syncthreads();
}


/*
 * slave thread will receive the partial XML file from master, and parse the xml into tags details,
 * including tag name, attribute,then uses these details to build tag invert index, which includes
 * the tag name as the key and the tag id as the value.
 * */
void* slave_parse(void* argc){

	MPI_Status s_prob,s_recv_txt,s_recv_info;
	MPI_Request request;
	size_t inverted_index_map_size = 134217728; //2^27
	tags_inverted_index = (hash_map*)malloc(sizeof(hash_map));
	hash_map_init(tags_inverted_index,inverted_index_map_size,MPQ_comparator,hash_map_MPQ_hash_func);
	prepareCryptTable();//CryptTable is for the MPQ hash algorithm

	char* bytes_of_text = (char*)malloc(sizeof(Text));
	memset(bytes_of_text,0,sizeof(Text));
	char* tag_info_bytes; //buffer for receiving Tags' information in the partial xml file
	tag_info_bytes =(char*)malloc(TAGS_PER_TIME*sizeof(tag_info));
	memset(tag_info_bytes,0,sizeof(tag_info)*TAGS_PER_TIME);

	int pf;
	pf = open(file_path,S_IRUSR);

	int base = 0;
	int slave_rank;
	int text_ready = 0, tag_info_ready = 0;
	int tags_num;
	MPI_Comm_rank(MPI_COMM_WORLD,&slave_rank);

	Text* t;
	tag_info* host_tag_infos;
	char* host_read_string;
	int l,r,ii=0;
	//receive the partial xml file and the structure informations of tags in this part of file
	while(1){
		MPI_Probe(0,MPI_ANY_TAG,MPI_COMM_WORLD,&s_prob);
		printf("slave receive a message %d\n",s_prob.MPI_TAG);
		int count;
		MPI_Get_count(&s_prob,MPI_CHAR,&count);
		switch(s_prob.MPI_TAG){
		case MSG_TEXT:
			MPI_Recv(bytes_of_text,count,MPI_CHAR,0,MSG_TEXT,MPI_COMM_WORLD,&s_recv_txt);
			t = (Text*)bytes_of_text;
			base = t->offset;
			printf("received text length is %d,offset is %d \n",t->length,t->offset);
			host_read_string = (char*)malloc(t->length);

		    l = lseek(pf,t->offset,SEEK_SET);
			if(l == -1) printf("lseek failed 000000000000000000000000000000\n");
			r = read(pf,host_read_string,t->length);
			if(r==-1) printf("read text failed 99999999999999999999999999999\n");
			else printf("receive text successfully, l:%d,r:%d\n",l,r);
			printf("CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC\n");
			printf("first 10 char of read text:%c,%c,%c,%c,%c,%c,%c,%c,%c,%c!!!\n",host_read_string[0],host_read_string[1],
					host_read_string[2],host_read_string[3],host_read_string[4],host_read_string[5],host_read_string[6],
					host_read_string[7],host_read_string[8],host_read_string[9]);
//			if(ii == 1) printf("%s\n",host_read_string);
			text_ready = 1;
//			printf("%s\n",host_read_string);

			break;
		case MSG_SEND_TAG_INFO:
			MPI_Recv(tag_info_bytes,count,MPI_CHAR,0,MSG_SEND_TAG_INFO,MPI_COMM_WORLD,&s_recv_info);
			tag_info_ready = 1;
			tags_num = count/sizeof(tag_info);
			host_tag_infos = (tag_info*)tag_info_bytes;
			printf("receive tag info successfully\n");
			printf("AAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAA\n");
			printf("slave tag 20's info is following:\n");
			printf("id:%d length:%d location:%d\n",host_tag_infos[20].id,host_tag_infos[20].lengh,host_tag_infos[20].location);
			break;
		case MSG_EXIT:
			int end;
//			MPI_Recv(&end,0,MPI_INT,0,MSG_EXIT,MPI_COMM_WORLD,&status);
			printf("EEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEE\n");
//			pthread_exit(NULL);
			return NULL;
//
		}


		if(text_ready && tag_info_ready){


			char* device_string_to_parse;
			Tag* device_tags;
			tag_info* device_tag_info;
			HANDLE_ERROR(hipMalloc((void**)&device_string_to_parse,t->length));
			HANDLE_ERROR(hipMemset(device_string_to_parse,0,t->length));
			HANDLE_ERROR(hipMalloc((void**)&device_tags,sizeof(Tag)*TAGS_PER_TIME));
			HANDLE_ERROR(hipMemset(device_tags,0,sizeof(Tag)*TAGS_PER_TIME));
			HANDLE_ERROR(hipMalloc((void**)&device_tag_info,sizeof(tag_info)*TAGS_PER_TIME));
			HANDLE_ERROR(hipMemset(device_tag_info,0,sizeof(tag_info)*TAGS_PER_TIME));
			HANDLE_ERROR(hipMemcpy(device_string_to_parse,host_read_string,t->length,hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(device_tag_info,host_tag_infos,sizeof(tag_info)*TAGS_PER_TIME,hipMemcpyHostToDevice));


//			printf("tags num is %d\n",tags_num);
			parse_tag<<<1,tags_num>>>(device_tag_info, device_string_to_parse,base,device_tags);

			Tag* host_tags_buffer = (Tag*)malloc(sizeof(Tag)*TAGS_PER_TIME);//buffer for restoring the tag parse result.
			memset(host_tags_buffer,0,sizeof(Tag)*TAGS_PER_TIME);
			HANDLE_ERROR(hipMemcpy(host_tags_buffer,device_tags,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyDeviceToHost));
			hipFree(device_string_to_parse);
			hipFree(device_tags);
			hipFree(device_tag_info);
			free(host_read_string);

			memset(tag_info_bytes,0,sizeof(tag_info)*TAGS_PER_TIME);
			printf("**********************************************************\n");
			printf("name: %s,id: %d\n",host_tags_buffer[20].name,host_tags_buffer[20].info.id);
			for(int i=0;i< TAGS_PER_TIME;i++) if(host_tags_buffer[i].nameCharIndex>0){

				char* key = host_tags_buffer[i].name;
//				printf("name :%s\n",key);
				int id = host_tags_buffer[i].info.id;
				hash_map_insert(tags_inverted_index,(void*)key,(void*)(&id));
			}
			free(host_tags_buffer);
			text_ready = 0;
			tag_info_ready = 0;
			idle = 1;
			//notify the master machine that this node has completed the partial parsing work
			MPI_Send(&idle,1,MPI_INT,0,MSG_IDLE,MPI_COMM_WORLD);
//			MPI_Wait(&request,&status);

			printf("BBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBBB\n");
		}
	}

//	pthread_t slave_query_thread;
////	int* temp_arg;
//	pthread_create(&slave_query_thread,NULL,slave_query_worker,NULL);
//	pthread_join(slave_query_thread,NULL);
//	return NULL;


}


char processor_name[MPI_MAX_PROCESSOR_NAME];

int main(int argc, char **argv) {
	char *filePath = argv[1];
	strcpy(file_path,filePath);
	simple_parse_arg *s_arg;
//
	int numprocs,namelen,rank,devCount, provide_level;
	MPI_Init_thread(&argc,&argv,MPI_THREAD_SINGLE,&provide_level);
	if(provide_level < MPI_THREAD_SINGLE){
		printf("Error: the MPI library doesn't provide the required thread level\n");
		MPI_Abort(MPI_COMM_WORLD,0);
	}
	MPI_Comm_size(MPI_COMM_WORLD,&numprocs);
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Get_processor_name(processor_name,&namelen);
	printf("Hello from %d on %s out of %d\n", rank, processor_name, numprocs);
	slaves_num = numprocs - 1;
	for(int i=0;i<5;i++)
			idle_node[i] = 1;
	printf("set slaves num %d \n",slaves_num);
	if(rank==0){
//		GDB_WAIT_ATTACH();
		//initial values
		printf("provide level is :%d\n",provide_level);
		file_read_over = 0;
		slave_parse_stop = 0;
		s_arg = (simple_parse_arg*)malloc(sizeof(simple_parse_arg));
		s_arg->file_path = filePath;
		s_arg->s_tags_ready = (tag_info**)malloc(sizeof(tag_info*));
		s_arg->text_read = (Text**)malloc(sizeof(Text*));
		*(s_arg->s_tags_ready) = (tag_info*)malloc(sizeof(tag_info));
		*(s_arg->text_read) = (Text*)malloc(sizeof(Text));
		pthread_t thread_prescan, thread_schedule;
		pthread_create(&thread_prescan,NULL,simple_parse,(void*)s_arg);
		pthread_create(&thread_schedule,NULL,schedule,(void*)s_arg);
		pthread_join(thread_prescan,NULL);
		pthread_join(thread_schedule,NULL);
	}
	else{
//		GDB_WAIT_ATTACH();
		if(hipGetDeviceCount(&devCount)!=hipSuccess){
			 printf("Device error on %s\n", processor_name);
			 MPI_Finalize();
			 return EXIT_FAILURE;
		}

		pthread_t slave_parse_thread;
		pthread_create(&slave_parse_thread,NULL,slave_parse,NULL);
		pthread_join(slave_parse_thread,NULL);

	}

	MPI_Finalize();
	return 0;
}
