#include "hip/hip_runtime.h"
#include "stack.h"
#include "tag.h"
#include "node.h"
#include "simple_state_machine.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include "simple_scan_parser.h"
#include "synchronize.h"
#define MAX_READ_LENGTH 1024*1024*8

#define TagNameAddChar(tag, c) do{			\
	tag->name[tag->nameCharIndex] = c;		\
	tag->nameCharIndex++;					\
}while(0)

#define TagAddAttr(tag, attr) do{			\
	tag->attributes[tag->attrIndex]=attr;	\
	tag->attrIndex++;						\
}while(0)

#define AttrNameAddChar(attr, c) do{		\
	attr->name[attr->nameCharIndex] = c;    \
	attr->nameCharIndex++;					\
}while(0)

#define AttrValueAddChar(attr,c) do{		\
	attr->value[attr->valueCharIndex] = c;  \
	attr->valueCharIndex++;					\
}while(0)

__global__ void parse_tag(Tag* tags_info, char* string){
	int id = threadIdx.x;
	Tag t = tags_info[id];
	int s_location = t.location;
	int len = t.lengh;
	
	int curIndex = s_location;
	int start = 0;
	int lastLTIndex = 0;
	int firstGTIndex = 0;
	enum _state st = st_idle;
	attribute *tmp_attr;
	while(curIndex - s_location < len){
		char cur = string[curIndex];

		switch(st){
			case st_idle:{
				//start state
				if('<' == cur){
					st = st_lt;
					break;
				}
				break;
			}//end of case st_idle
			case st_lt:{	
				if('>' == cur){
					st = st_idle;
					break;
				}
				//if cur is other character, this node is a start tag
				st = st_start_tag;
				break;
			}//end of state tagstart
			case st_start_tag:{//record tag name here
				if(' ' == cur){
					tmp_attr = (attribute*)malloc(sizeof(attribute));
					st = st_attribute_pre;
				}else if('>' == cur){
					st = st_idle;
				}else{// else this char belongs to tag name
					TagNameAddChar((&t),cur);
				}
				break;
			}//end of state st_start_tag
			case st_attribute_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){// normal end
					st = st_idle;
				}else{
					AttrNameAddChar((&t),cur);
					st = st_attribute_name;
				}
				break;
			}
			case st_attribute_name:{
				if('>' == cur){
					st = st_idle;
				}else if('=' == cur){
					st = st_attribute_value_pre;
				}else{
					AttrNameAddChar((&t),cur);
				}
				break;
			}
			case st_attribute_value_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){
					st = st_idle;
				}else if(('"'==cur)||('\''==cur)){
					st = st_attribute_value;
				}
				break;
			}
			case st_attribute_value:{
				if(('"'==cur)||('\''==cur)){
					TagAddAttr((&t),*tmp_attr);
					free(tmp_attr);
					st = st_attribute_pre;
				}else{
					AttrValueAddChar(tmp_attr,cur);
				}
				break;
			}

		}//end of switch
		
		curIndex ++;
		continue;
		done:
		break;
	}//end of read buffer.

}


void cuda_parse(char *host_read_string, Tag *host_tags_info){
	while(1){
		syn_suspend(cond_cuda);
		char* device_string_to_parse;
		Tag* device_tags_info;

		size_t host_string_len = strlen(host_read_string);
		hipMalloc((void**)&device_string_to_parse,sizeof(char)*host_string_len);
		hipMalloc((void**)&device_tags_info,sizeof(Tag)*TAGS_PER_TIME);
		hipMemcpy(device_string_to_parse,host_read_string,host_string_len,hipMemcpyHostToDevice);
		hipMemcpy(device_tags_info,host_tags_info,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyHostToDevice);
		parse_tag<<<1,1024>>>(device_tags_info, device_string_to_parse);
		hipMemcpy(host_tags_info,device_tags_info,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyDeviceToHost);
		hipFree(device_string_to_parse);
		hipFree(device_tags_info);
		syn_resume(&cond_pre);
	}
}


int main(int argc, char **argv) {

	syn_init();

	pthread_t thread_prescan, thread_cuda_parse;

	char filePath[1000]="/home/jerry/Downloads/enwiki-latest-pages-meta-history4.xml-p000104986p000104998";//80.9M
	FILE *file;
	file = fopen(filePath,"r");
	if(!file)
	{
		printf("open file failed \n");
		return -1;
	}
	char* host_buffer;
	Tag* host_tags_info;
	char* host_read_string;
	simple_parse(file,host_buffer,host_tags_info,host_read_string);
	

	pthread_create(&thread_prescan,NULL,simple_parse(file,host_buffer,host_tags_info,host_read_string),(void*)&argc);
	pthread_create(&thread_cuda_parse,NULL,cuda_parse(host_read_string,host_tags_info),(void*)&argc);

	pthread_join(thread_prescan);
	pthread_join(thread_cuda_parse);

}
