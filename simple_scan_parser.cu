#include "hip/hip_runtime.h"

#include "synchronize.h"
#include "simple_state_machine.h"
#include "stack.h"
#include "tag.h"
#include "node.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include "simple_scan_parser.h"
#include <time.h>
#include <pthread.h>
#include <unistd.h>
#define MAX_READ_LENGTH 1024*1024*8

#define TagNameAddChar(tag, c) do{			\
	tag->name[tag->nameCharIndex] = c;		\
	tag->nameCharIndex++;					\
}while(0)

#define TagAddAttr(tag, attr) do{			\
	tag->attributes[tag->attrIndex]=attr;	\
	tag->attrIndex++;						\
}while(0)

#define AttrNameAddChar(attr, c) do{		\
	attr->name[attr->nameCharIndex] = c;    \
	attr->nameCharIndex++;					\
}while(0)

#define AttrValueAddChar(attr,c) do{		\
	attr->value[attr->valueCharIndex] = c;  \
	attr->valueCharIndex++;					\
}while(0)

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//__constant__ unsigned char pBuffer[16*16*16*16*LENGTH];

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

typedef struct{
	char **host_read_string;
	Tag **host_tags_info;
}cuda_arg;

extern "C"{
	void syn_init();
	void syn_suspend(pthread_cond_t cond);
	void syn_resume(pthread_cond_t cond);
	void* simple_parse(void* arg);
}
__global__ void parse_tag(Tag* tags_info, char* string){
	int id = threadIdx.x;
	Tag t = tags_info[id];
	int s_location = t.location;
	int len = t.lengh;
	
	int curIndex = s_location;
	enum _state st = st_idle;
	attribute tmp_attr;
	while(curIndex - s_location < len){
		char cur = string[curIndex];

		switch(st){
			case st_idle:{
				//start state
				if('<' == cur){
					st = st_lt;
					break;
				}
				break;
			}//end of case st_idle
			case st_lt:{	
				if('>' == cur){
					st = st_idle;
					break;
				}
				//if cur is other character, this node is a start tag
				st = st_start_tag;
				break;
			}//end of state tagstart
			case st_start_tag:{//record tag name here
				if(' ' == cur){
					memset(&tmp_attr,0,sizeof(attribute));
					st = st_attribute_pre;
				}else if('>' == cur){
					st = st_idle;
				}else{// else this char belongs to tag name
					TagNameAddChar((&t),cur);
				}
				break;
			}//end of state st_start_tag
			case st_attribute_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){// normal end
					st = st_idle;
				}else{
					AttrNameAddChar((&t),cur);
					st = st_attribute_name;
				}
				break;
			}
			case st_attribute_name:{
				if('>' == cur){
					st = st_idle;
				}else if('=' == cur){
					st = st_attribute_value_pre;
				}else{
					AttrNameAddChar((&t),cur);
				}
				break;
			}
			case st_attribute_value_pre:{
				if(' ' == cur){
					//ignore space
				}else if('>' == cur){
					st = st_idle;
				}else if(('"'==cur)||('\''==cur)){
					st = st_attribute_value;
				}
				break;
			}
			case st_attribute_value:{
				if(('"'==cur)||('\''==cur)){
					TagAddAttr((&t),tmp_attr);
					//free(tmp_attr);
					st = st_attribute_pre;
				}else{
					AttrValueAddChar((&tmp_attr),cur);
				}
				break;
			}

		}//end of switch
		
		curIndex ++;
		continue;
		done:
		break;
	}//end of read buffer.

}

void* cuda_parse(void* c_arg){

	while(1){
//		printf("cuda_parse start...... \n");
		pthread_cond_wait(&cond_cuda,&syn_mutex);
		printf("cuda receive signal\n");
		simple_parse_arg *arg = (simple_parse_arg*)c_arg;
		char *host_read_string = *(arg->string_read);
		Tag *host_tags_info = *(arg->s_tags_ready);
		char* device_string_to_parse;
		Tag* device_tags_info;

		size_t host_string_len = strlen(host_read_string);
		hipMalloc((void**)&device_string_to_parse,sizeof(char)*host_string_len);
		hipMalloc((void**)&device_tags_info,sizeof(Tag)*TAGS_PER_TIME);
		hipMemcpy(device_string_to_parse,host_read_string,host_string_len,hipMemcpyHostToDevice);
		hipMemcpy(device_tags_info,host_tags_info,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyHostToDevice);
		parse_tag<<<1,1024>>>(device_tags_info, device_string_to_parse);
		hipMemcpy(host_tags_info,device_tags_info,sizeof(Tag)*TAGS_PER_TIME,hipMemcpyDeviceToHost);
		hipFree(device_string_to_parse);
		hipFree(device_tags_info);
		pthread_cond_signal(&cond_prescan);
		printf("cuda send signal \n");
	}
}





int main(int argc, char **argv) {

//
	pthread_t thread_prescan, thread_cuda_parse;
	pthread_t thread_resume;
//	char filePath[1000]="/home/jerry/Downloads/enwiki-latest-pages-meta-history4.xml-p000104986p000104998";//80.9M
	char *filePath = argv[1];
	Tag* host_tags_info;
	char* host_read_string;
	simple_parse_arg *s_arg;
	s_arg = (simple_parse_arg*)malloc(sizeof(simple_parse_arg));
	s_arg->file_path = filePath;
	s_arg->s_tags_ready = (Tag**)malloc(sizeof(Tag*));
	s_arg->string_read = (char**)malloc(sizeof(char*));
	*(s_arg->s_tags_ready) = (Tag*)malloc(sizeof(Tag));
	*(s_arg->string_read) = (char*)malloc(sizeof(char));

	cuda_arg c_arg;
	clock_t start,stop;
	start = clock();



	pthread_create(&thread_cuda_parse,NULL, cuda_parse,(void*)s_arg);
	pthread_create(&thread_prescan,NULL,simple_parse,(void*)s_arg);
	pthread_join(thread_cuda_parse,NULL);
	pthread_join(thread_prescan,NULL);
//	stop = clock();
//	double dur = (double)(stop - start);
//	printf("\n total time is: %f s\n",dur/CLOCKS_PER_SEC);
	pthread_exit(NULL);

}
